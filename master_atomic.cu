#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.cuh"

__global__ void incr(int *ptr)
{
    /*
    int temp = *ptr;
    temp = temp + 1;
    *ptr = temp;
    */
    atomicAdd(ptr, 1);
}

__device__ int myAtomicAdd(int *address, int incr)
{
    int expected = *address;
    int oldValue = atomicCAS(address, expected, expected + incr);

    while(oldValue != expected)
    {
        expected = oldValue;
        oldValue = atomicCAS(address, expected, expected + incr);
    }
    return oldValue;
}

__global__ void new_atomic_add_test(int *ptr)
{
    myAtomicAdd(ptr,1);
}

void intro_newadd()
{
	int value = 0;
	int SIZE = sizeof(int);
	int ref = -1;

	int *d_val;
	hipMalloc((void**)&d_val, SIZE);
	hipMemcpy(d_val, &value, SIZE, hipMemcpyHostToDevice);
	incr << <1, 32 >> > (d_val);
    new_atomic_add_test << <1, 32 >> > (d_val);
	hipDeviceSynchronize();
	hipMemcpy(&ref,d_val,SIZE, hipMemcpyDeviceToHost);

	printf("Updated value : %d \n",ref);

	hipDeviceReset();
}

__global__ void atomics(int *shared_var, int iters)
{
    for (int i = 0; i < iters; i++)
    {
        atomicAdd(shared_var, 1);
    }
}

__global__ void unsafe(int *shared_var, int iters)
{
    for (int i = 0; i < iters; i++)
    {
        int old = *shared_var;
        *shared_var = old + 1;
    }
}

void atomic_performance()
{
	int N = 64;
	int block = 32;
	int runs = 30;
	int iters = 100000;
	int r;
	int *d_shared_var;
	int h_shared_var_atomic, h_shared_var_unsafe;
	int *h_values_read;

	gpuErrchk(hipMalloc((void **)&d_shared_var, sizeof(int)));

	double atomic_mean_time = 0;
	double unsafe_mean_time = 0;
	clock_t ops_start, ops_end;

	for (r = 0; r < runs; r++)
	{
		gpuErrchk(hipMemset(d_shared_var, 0x00, sizeof(int)));

		ops_start = clock();
		atomics <<< N / block, block >>>(d_shared_var,iters);
		gpuErrchk(hipDeviceSynchronize());
		ops_end = clock();
		atomic_mean_time += ops_end - ops_start;

		gpuErrchk(hipMemcpy(&h_shared_var_atomic, d_shared_var, sizeof(int),
			hipMemcpyDeviceToHost));
		gpuErrchk(hipMemset(d_shared_var, 0x00, sizeof(int)));

        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start);
        //ops_start = clock();
		unsafe <<< N / block, block >>>(d_shared_var,iters);
        hipEventRecord(end);
        hipEventSynchronize(end);
        float time;
        hipEventElapsedTime(&time, start, end);
        //printf("Kernel execution time using events : %f \n",time);
		gpuErrchk(hipDeviceSynchronize());
		//ops_end = clock();
		//unsafe_mean_time += ops_end - ops_start;
        unsafe_mean_time += time;

                gpuErrchk(hipMemcpy(&h_shared_var_unsafe, d_shared_var, sizeof(int),
			hipMemcpyDeviceToHost));
	}

	atomic_mean_time = atomic_mean_time / CLOCKS_PER_SEC;
	//unsafe_mean_time = unsafe_mean_time / CLOCKS_PER_SEC;

	printf("In total, %d runs using atomic operations took %f s\n",
		runs, atomic_mean_time);
	printf("  Using atomic operations also produced an output of %d\n",
		h_shared_var_atomic);
	printf("In total, %d runs using unsafe operations took %f s\n",
		runs, unsafe_mean_time);
	printf("  Using unsafe operations also produced an output of %d\n",
		h_shared_var_unsafe);

}

int main()
{
    //intro_newadd();
    atomic_performance();
    return 0;
}