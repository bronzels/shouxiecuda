#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include "common.hpp"

#include <stdio.h>
#include <ctime>
#include <random>
using namespace std;

__global__ void stream_test(int* in, int* out, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < size)
    {
        for (int i = 0; i < 25; i ++)
        {
            out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
        }
    }
}

int async()
{
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    float gpu_time = 0.0f;
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    int size = 1 << 26;
    int byte_size = size * sizeof(int);

    //host pointers
    int *h_in, *h_ref, *h_in2, *h_ref2;

    //allocate memory for host pointers
    hipHostMalloc((void **)&h_in, byte_size);
    hipHostMalloc((void **)&h_ref, byte_size);
    hipHostMalloc((void **)&h_in2, byte_size);
    hipHostMalloc((void **)&h_ref2, byte_size);

    initialize(h_in, size, INIT_ONE_TO_TEN);
    initialize(h_in2, size, INIT_ONE_TO_TEN);

    //device pointers
    int *d_in, *d_out, *d_in2, *d_out2;

    //allocate memory for host pointers
    hipMalloc((void **)&d_in, byte_size);
    hipMalloc((void **)&d_out, byte_size);
    hipMalloc((void **)&d_in2, byte_size);
    hipMalloc((void **)&d_out2, byte_size);

    hipStream_t str, str2;
    hipStreamCreate(&str);
    hipStreamCreate(&str2);

    //luanching the grid
    dim3 block(128);
    dim3 grid((size + block.x - 1) / block.x);

    hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice, str);
    stream_test <<< grid, block>>> (d_in, d_out, size);
    hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost, str);

    hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
    stream_test <<< grid, block>>> (d_in2, d_out2, size);
    hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);

    hipStreamSynchronize(str);
    hipStreamDestroy(str);

    hipStreamSynchronize(str2);
    hipStreamDestroy(str2);

    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    unsigned long int counter = 0;
    while(hipEventQuery(stop) == hipErrorNotReady)
    {
        counter ++;
    }
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("async time spent executing by the GPU: %.6f\n", gpu_time);

    compare_arrays(h_ref, h_ref2, size);

    hipHostFree(h_in);
    hipHostFree(h_in2);
    hipHostFree(h_ref);
    hipHostFree(h_ref2);
    hipFree(d_in);
    hipFree(d_in2);
    hipFree(d_out);
    hipFree(d_out2);

    checkCudaErrors(hipDeviceReset());
    return 0;
}

int sync()
{
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    float gpu_time = 0.0f;
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    int size = 1 << 27;
    int byte_size = size * sizeof(int);

    //host pointers
    int *h_in, *h_ref;

    //allocate memory for host pointers
    h_in = (int *)malloc(byte_size);
    h_ref = (int *)malloc(byte_size);

    initialize(h_in, size, INIT_ONE_TO_TEN);

    //device pointers
    int *d_in, *d_out;

    //allocate memory for host pointers
    checkCudaErrors(hipMalloc((void **)&d_in, byte_size));
    checkCudaErrors(hipMalloc((void **)&d_out, byte_size));

    //luanching the grid
    dim3 block(128);
    dim3 grid((size + block.x - 1) / block.x);

    checkCudaErrors(hipMemcpy(d_in, h_in, byte_size, hipMemcpyHostToDevice));
    stream_test <<< grid, block>>> (d_in, d_out, size);
    checkCudaErrors(hipMemcpy(h_ref, d_out, byte_size, hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    unsigned long int counter = 0;
    while(hipEventQuery(stop) == hipErrorNotReady)
    {
        counter ++;
    }
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("sync time spent executing by the GPU: %.6f\n", gpu_time);

    free(h_in);
    free(h_ref);
    checkCudaErrors(hipFree(d_in));
    (hipFree(d_out));

    checkCudaErrors(hipDeviceReset());
    return 0;
}

/*
int size = 1 << 19
async time spent executing by the GPU: 1.675520
Arrays are same
sync time spent executing by the GPU: 1.949344

int size = 1 << 22
async time spent executing by the GPU: 12.006400
Arrays are same
sync time spent executing by the GPU: 11.176992


int size = 1 << 27
async time spent executing by the GPU: 366.351166
Arrays are same
sync time spent executing by the GPU: 337.174377

 */

int main(int argc, char** argv) {
    async();
    sync();
}