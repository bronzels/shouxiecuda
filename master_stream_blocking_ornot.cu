#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void blocking_nonblocking_testnull()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid == 0)
    {
        for (size_t i = 0; i < 10000; i++)
        {
            printf("stream null \n");
        }
    }
}

__global__ void blocking_nonblocking_test1()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid == 0)
    {
        for (size_t i = 0; i < 10000; i++)
        {
            printf("stream 1 \n");
        }
    }
}

__global__ void blocking_nonblocking_test2()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid == 0)
    {
        for (size_t i = 0; i < 10000; i++)
        {
            printf("stream 2 \n");
        }
    }
}

__global__ void blocking_nonblocking_test3()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid == 0)
    {
        for (size_t i = 0; i < 10000; i++)
        {
            printf("stream 3 \n");
        }
    }
}

__global__ void blocking_nonblocking_test4()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid == 0)
    {
        for (size_t i = 0; i < 10000; i++)
        {
            printf("stream 4 \n");
        }
    }
}

int main(int argc, char ** argv)
{
	int size = 1 << 15;

    hipStream_t stm1, stm2, stm3, stm4;

    hipEvent_t event1;
    hipEventCreate(&event1, hipEventDisableTiming);

    hipStreamCreateWithFlags(&stm1, hipStreamNonBlocking);
    hipStreamCreate(&stm2);
    hipStreamCreateWithFlags(&stm3, hipStreamNonBlocking);
    hipStreamCreate(&stm4);

	dim3 block(128);
	dim3 grid(size / block.x);



    blocking_nonblocking_test1<<<grid, block, 0, stm1>>>();
    hipEventRecord(event1, stm1);
    hipStreamWaitEvent(stm3, event1, 0);

    blocking_nonblocking_test2<<<grid, block, 0, stm2>>>();
    blocking_nonblocking_testnull<<<grid, block>>>();
    blocking_nonblocking_test3<<<grid, block, 0, stm3>>>();
    blocking_nonblocking_test4<<<grid, block, 0, stm4>>>();

    hipEventDestroy(event1);
/*
stream 2
stream 3
stream 1
stream 2
stream 3
stream 1
stream 2
stream 3
stream 1
stream 2
stream 3
stream 1
stream 2
stream 3
stream null
stream null
stream null
stream null
stream null
stream null
stream null
stream null
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
*/

/*hipStreamWaitEvent(stm3, event1, 0);
stream 1
stream 2
stream 1
stream 2
stream 1
stream 2
stream 1
stream 2
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 3
stream null
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
stream 4
*/
    hipStreamDestroy(stm1);
    hipStreamDestroy(stm2);
    hipStreamDestroy(stm3);
    hipStreamDestroy(stm4);

	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipDeviceReset());
	return 0;
}