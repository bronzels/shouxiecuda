#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.cpph"
#include "cuda_common.cuh"

#include "hip/hip_runtime_api.h"
#include "helper_functions.h"

#include <ctime>
#include <random>
#include <time.h>
using namespace std;

__global__ void gpuRecursiveReduce(int * g_idata,
                                                int * g_odata, unsigned int isize)
{
    int tid = threadIdx.x;

    int * idata = g_idata + blockIdx.x * blockDim.x;
    int * odata = &g_odata[blockIdx.x];

    //return condition
    if(isize == 2 && tid == 0)
    {
        g_odata[blockIdx.x] = idata[0] + idata[1];
        return;
    }

    int istride = isize >> 1;

    if (istride > 1 && tid < istride)
    {
        idata[tid] += idata[tid + istride];
    }

    //__syncthreads();

    if(tid == 0)
    {
        gpuRecursiveReduce <<<1, istride>>>(idata, odata, istride);
        //hipDeviceSynchronize();
    }

    //__syncthreads();
}

/*
2sync-47.82 msec
nosync-48.27 msec
*/
/*
                        time spent executing by the GPU      time spent by CPU in CUDA calls
2sync                   33.398079                            0.023000
nosync                  34.398079                            0.023000
*/
int main(int argc, char ** argv)
{
	printf("Running parallel reduction with interleaved pairs kernel \n");

	int size = 1 << 22;
	int byte_size = size * sizeof(int);
	int block_size = 512;
	clock_t gpu_start, gpu_end,cpu_start, cpu_end;

	int * h_input, *h_ref;
	h_input = (int*)malloc(byte_size);
	initialize(h_input, size, INIT_RANDOM);

	cpu_start = clock();
	int cpu_result = reduction_cpu(h_input, size);
	cpu_end = clock();

	dim3 block(block_size);
	dim3 grid(size / block.x);

	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);

	int temp_array_byte_size = sizeof(int)* grid.x;

	h_ref = (int*)malloc(temp_array_byte_size);

	int * d_input, *d_temp;
	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	gpu_start = clock();

	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
		hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    float gpu_time = 0.0f;
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
	gpuRecursiveReduce <<< grid, block >>> (d_input, d_temp,block_size);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    unsigned long int counter = 0;
    while(hipEventQuery(stop) == hipErrorNotReady)
    {
        counter ++;
    }
    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
    printf("time spent executing by the GPU: %.6f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.6f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}
	gpu_end = clock();
	print_time_using_host_clock(gpu_start, gpu_end);

	printf("CPU kernel execution time : %4.6f \n",
		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));

	compare_results(gpu_result, cpu_result);

	gpuErrchk(hipFree(d_input));
	gpuErrchk(hipFree(d_temp));
	free(h_input);
	free(h_ref);

	hipDeviceReset();
	return 0;
}