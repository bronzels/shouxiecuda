#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void mem_trs_test(int size, int * input)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if(gid < size)
        printf("tid : %d, gid : %d, value : %d \n", threadIdx.x,gid,input[gid]);
}

int main()
{
    int size = 150;
    int byte_size = sizeof(int) * size;

    int * h_input;
    h_input = (int *)malloc(byte_size);

    time_t t;
    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++)
    {
        h_input[i] = (int)(rand() & 0xff);
    }


    int * d_input;
    hipMalloc((void **)&d_input, byte_size);
    hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

    dim3 block(32);
    dim3 grid(5);
    mem_trs_test <<< grid, block >>> (size, d_input);
    hipDeviceSynchronize();

    free(h_input);
    hipFree(d_input);
    hipDeviceReset();
    return 0;
}