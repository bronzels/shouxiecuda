#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>
#include <thrust/equal.h>
#include <thrust/iterator/constant_iterator.h>
#include <hipblas.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <cstdlib>
#define USECPSEC 1000000ULL

long long dtime_usec(unsigned long long start){

    timeval tv;
    gettimeofday(&tv, 0);
    return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

struct dp
{
    float *A, *B;
    int m,n,r;
    dp(float *_A, float *_B, int _m, int _n, int _r): A(_A), B(_B), m(_m), n(_n), r(_r) {};
    __host__ __device__
    float operator()(size_t idx){
        float sum = 0.0f;
        int row = idx/r;
        int col = idx - (row*r); // cheaper modulo
        for (int i = 0; i < m; i++)
            sum += A[col + row*i] * B[col + row*i];
        return sum;}
};

const int dsd = 200;
int main(int argc, char *argv[]){
    int ds = dsd;
    if (argc > 1) ds = atoi(argv[1]);
    const int n = ds;
    const int m = ds;
    const int r = ds;
    // data setup
    thrust::device_vector<float> data(n*m,1);
    thrust::device_vector<float> other(m*r,1);
    thrust::device_vector<float> result(n*r,0);
    // method 1
    //let's pretend that other is (already) transposed for efficient memory access by thrust
    // therefore each dot-product is formed using a row of data and a row of other
    long long dt = dtime_usec(0);
    if (ds < 201){
        for (int i = 0; i < n; ++i)
        {
            for (int j = 0; j < r;++j)
            {
                result[i*r+ j] = thrust::inner_product(data.begin()+(i*m), data.begin()+((i+1)*m),other.begin()+(j*m), 0.0f);
            }
        }
        hipDeviceSynchronize();
        dt = dtime_usec(dt);
        if (thrust::equal(result.begin(), result.end(), thrust::constant_iterator<float>(m)))
            std::cout << "method 1 time: " << dt/(float)USECPSEC << "s" << std::endl;
        else
            std::cout << "method 1 failure" << std::endl;
    }
    thrust::fill(result.begin(), result.end(), 0);
    hipDeviceSynchronize();
// method 2
    //let's pretend that data is (already) transposed for efficient memory access by thrust
    // therefore each dot-product is formed using a column of data and a column of other
    dt = dtime_usec(0);
    thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(n*r), result.begin(), dp(thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(other.data()), m, n, r));
    hipDeviceSynchronize();
    dt = dtime_usec(dt);
    if (thrust::equal(result.begin(), result.end(), thrust::constant_iterator<float>(m)))
        std::cout << "method 2 time: " << dt/(float)USECPSEC << "s" << std::endl;
    else
        std::cout << "method 2 failure" << std::endl;
// method 3
    // once again, let's pretend the data is ready to go for CUBLAS
    hipblasHandle_t h;
    hipblasCreate(&h);
    thrust::fill(result.begin(), result.end(), 0);
    float alpha = 1.0f;
    float beta = 0.0f;
    hipDeviceSynchronize();
    dt = dtime_usec(0);
    hipblasSgemm(h, HIPBLAS_OP_T, HIPBLAS_OP_T, n, r, m, &alpha, thrust::raw_pointer_cast(data.data()), n, thrust::raw_pointer_cast(other.data()), m, &beta, thrust::raw_pointer_cast(result.data()), n);
    hipDeviceSynchronize();
    dt = dtime_usec(dt);
    if (thrust::equal(result.begin(), result.end(), thrust::constant_iterator<float>(m)))
        std::cout << "method 3 time: " << dt/(float)USECPSEC << "s" << std::endl;
    else
        std::cout << "method 3 failure" << std::endl;
}
